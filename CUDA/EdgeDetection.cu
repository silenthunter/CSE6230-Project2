#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include "bmpLoader.cu"

//Program constants
const int cuda_threadX = 32;
const int cuda_threadY = 16;
const int gaussSize = 3;

__device__ const float lowThreshold = .1f;
__device__ const float  highThreshold = .3f;

__device__ float e = 2.71828f;
__device__ float PI = 3.14f;
__device__ float dev = 1.4f;
__device__ float *GaussMat;
__device__ int gaussWidth;
__device__ int halfGauss;
__device__ float *image;
__device__ float *imageBuf;
__device__ float *angles;
__device__ int width;
__device__ int height;
__device__ float Kgx[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
__device__ float Kgy[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
extern __shared__ float shared[];

__device__ int GetIdx()
{
	int idx = (threadIdx.x + blockDim.x * blockIdx.x) + (threadIdx.y + blockIdx.y * blockDim.y) *
		blockDim.x * gridDim.x;
	return idx;
}

__device__ int GetIdxBlock()
{
	int idx = (threadIdx.x + threadIdx.y * blockDim.x) +
		(blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y);
	return idx;
}

__device__ float GetGaussVal(int x, int y)
{
	x += gaussWidth / 2;
	y += gaussWidth / 2;
	return GaussMat[x + y * gaussWidth];
}

__device__ float GetSobelValX(int x, int y)
{
	x += 1;
	y += 1;
	return Kgx[x + y * 3];
}

__device__ float GetSobelValY(int x, int y)
{
	x += 1;
	y += 1;
	return Kgy[x + y * 3];
}

__device__ float GetPixel(int cur, int x = 0, int y = 0)
{
	cur = threadIdx.x + (threadIdx.y + halfGauss) * (blockDim.x + halfGauss * 2) + halfGauss;
	int idx = cur + x + y * (blockDim.x + halfGauss * 2);

	return shared[idx];
}

__device__ float GetPixelGlobal(int cur, int x = 0, int y = 0)
{
	if((int)(threadIdx.x) - x < 0 && blockIdx.x == 0) return imageBuf[cur];
	if(/*threadIdx.x + x >= blockDim.x ||*/ blockIdx.x >= gridDim.x) return imageBuf[cur];

	if((int)(threadIdx.y) - y < 0 && blockIdx.y == 0) return imageBuf[cur];
	if(/*threadIdx.y + y >= blockDim.y ||*/ blockIdx.y >= gridDim.y) return imageBuf[cur];

	int idx = cur + x + y * width;
	return imageBuf[idx];
}

__device__ float GaussianBlur(int x, int y)
{
	float p1 = 1.0f / (2 * PI * powf(dev, 2));
	float p2 = p1 * powf(e, -(powf(x, 2) + powf(y, 2)) / (2 * powf(dev, 2)));
	return p2;
}

__global__ void MakeBW(float* d_image, float* d_bw, float* d_buf, float* d_angles, int d_width, int d_height)
{	
	//TODO: Make local shared array to maximize speed
	int dest = GetIdx();
	int idx = dest * 3;//3 color values

	//http://www.bobpowell.net/grayscale.htm
	float val = d_image[idx] *.3 + d_image[idx + 1] * .59 + d_image[idx + 2] * .11;
	
	//Have thread 0 assign the global variables
	if(dest == 0)
	{
		image = d_bw;
		width = d_width;
		height = d_height;
		imageBuf = d_buf;
		angles = d_angles;
	}

	d_bw[dest] = val;
	d_buf[dest] = val;
}

__global__ void ComputeGuassian(float* d_gaussMat, int width)
{
	__shared__ float matSum;
	int idx = threadIdx.x + threadIdx.y * width;
	if(idx == 0)
	{
		gaussWidth = width;
		halfGauss = width / 2;
		GaussMat = d_gaussMat;
	}
	float val = GaussianBlur(threadIdx.x - width / 2, threadIdx.y - width / 2);
	d_gaussMat[idx] = val;
	syncthreads();
	
	//Have the first thread compute the sum
	//TODO: make this parallel?
	if(idx == 0)
	{
		matSum = .01;//Starts the sum with a small value to avoid sums > 1.0
		for(int i = 0; i < width * width; i++)
				matSum += d_gaussMat[i];
	}
	syncthreads();
	d_gaussMat[idx] = val / matSum;
}

__device__ void CopyToShared()
{
	int idx = GetIdx();
	int lWidth = 2 * halfGauss + blockDim.x;
	int localIdx = threadIdx.x + (threadIdx.y + halfGauss) * (blockDim.x + halfGauss * 2) + halfGauss;
	
	shared[localIdx] = imageBuf[idx];
	
	/*if(threadIdx.x == 0 && threadIdx.y == 0)
	{
		for(int i = -halfGauss; i < (int)blockDim.x + halfGauss; i++)
		{
			for(int j = -halfGauss; j < (int)blockDim.y + halfGauss; j++)
			{
				//shared[localIdx + i + j * lWidth] = imageBuf[idx + i + j * width];
				shared[localIdx + j * lWidth + i] = imageBuf[idx + j * width + i];
			}
		}
	}*/
	
	//Corners
	//UL
	if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x != 0 && blockIdx.y != 0)
	{
		//for(int i = -halfGauss; i <= 0; i++)
			//shared[localIdx + i + i * lWidth] = imageBuf[idx + i + i * width];
		shared[0] = imageBuf[idx - 1 - width];
	}
	//BR
	if(threadIdx.y == blockDim.y - 1 && threadIdx.x == blockDim.x - 1
		&& blockIdx.x != gridDim.x - 1  && blockIdx.y != gridDim.y - 1)
	{
		for(int i = 1; i <= halfGauss; i++)
			shared[localIdx + i + i * lWidth] = imageBuf[idx + i + i * width];
	}
	//UR
	if(threadIdx.y == 0 && threadIdx.x == blockDim.x - 1 &&
		blockIdx.y != 0 && blockIdx.x != gridDim.x - 1)
	{
		for(int i = 1, j = -1; i <= halfGauss; i++, j--)
			shared[localIdx + i + j * lWidth] = imageBuf[idx + i + j * width];
	}
	//BL
	if(threadIdx.y == blockDim.y - 1 && threadIdx.x == 0 &&
		blockIdx.y != gridDim.y - 1 && blockIdx.x != 0)
	{
		for(int i = -halfGauss, j = halfGauss; i <= 0; i++, j--)
			shared[localIdx + i + j * lWidth] = imageBuf[idx + i + j * width];
	}
	
	//Edges
	if(threadIdx.x == 0 && blockIdx.x != 0)
	{
		for(int i = -halfGauss; i < 0; i++) shared[localIdx + i] = imageBuf[idx + i];
	}
	else if(threadIdx.y == 0 && blockIdx.y != 0)
	{
		for(int i = -halfGauss; i < 0; i++) shared[localIdx + i * lWidth] = imageBuf[idx + i * width];
	}
	else if(threadIdx.y == blockDim.y - 1 && blockIdx.y != gridDim.y - 1)
	{
		for(int i = 1; i <= halfGauss; i++) shared[localIdx + i * lWidth] = imageBuf[idx + i * width];
	}
	else if(threadIdx.x == blockDim.x - 1 && blockIdx.x != gridDim.x - 1)
	{
		for(int i = 1; i <= halfGauss; i++) shared[localIdx + i] = imageBuf[idx + i];
	}
	syncthreads();
}

__global__ void GaussianBlur()
{
	int i, j;
	int idx = GetIdx();
	CopyToShared();
	float val = 0;//GetPixel(idx, 0, 0) * .999f;
	
 	for( i = -halfGauss; i <= halfGauss; i++)
	{
		for( j = -halfGauss; j <= halfGauss; j++)
		{
			val += GetPixel(idx, i, j) * GetGaussVal(i, j);
		}
	}
	
	image[idx] = val;
	
}

__global__ void CopyToBuffer()
{
	int idx = GetIdx();
	imageBuf[idx] = image[idx];
}

__global__ void FindGradient()
{
	int i, j;
	int idx = GetIdx();
	float Gx = 0, Gy = 0;
	
	CopyToShared();
	
 	for( i = -1; i <= 1; i++)
	{
		for( j = -1; j <= 1; j++)
		{
			Gx += GetPixel(idx, i, j) * GetSobelValX(i, j);
			Gy += GetPixel(idx, i, j) * GetSobelValY(i, j);
		}
	}
	
	syncthreads();
	image[idx] = sqrt(powf(Gx, 2) + powf(Gy, 2));
	angles[idx] = atanf(abs(Gy) / abs(Gx));
}

__global__ void Suppression()
{
	const float step = PI / 4;
	int idx = GetIdxBlock();
	int count = 0;//Use an int to store angle. Better for comparison than float
	int angle = 0;
	float imgBuf = GetPixelGlobal(idx);
	
	for(float i = -PI / 2; i < PI / 2; i += step, count++)
	{
		if(angles[idx] - i < step / 2)
		{
			angle = count;
			angles[idx] = count;
			break;
		}
	}
	
	if(angle == 2)// Up and down
	{
		if(GetPixelGlobal(idx, 0, 1) > imgBuf || GetPixelGlobal(idx, 0, -1) > imgBuf)
			image[idx] = 0;
		else
			image[idx] = imgBuf;
	}
	else if(angle == 3) // UR and DL
	{
		if(GetPixelGlobal(idx, 1, 1) > imgBuf || GetPixelGlobal(idx, -1, -1) > imgBuf)
			image[idx] = 0;
		else
			image[idx] = imgBuf;
	}
	else if(angle == 0) // Left and Right
	{
		if(GetPixelGlobal(idx, 1, 0) > imgBuf || GetPixelGlobal(idx, -1, 0) > imgBuf)
			image[idx] = 0;
		else
			image[idx] = imgBuf;
	}
	else if(angle == 1) // UL and DR
	{
		if(GetPixelGlobal(idx, -1, 1) > imgBuf || GetPixelGlobal(idx, 1, -1) > imgBuf)
			image[idx] = 0;
		else
			image[idx] = imgBuf;
	}
	
	if(image[idx] > highThreshold) image[idx] = 0.95f;
	else if (image[idx] > lowThreshold) image[idx] = .5f;
	else image[idx] = 0;
}

__global__ void hysteresis()
{
	int idx = GetIdx();
	image[idx] = 0;
	
	if(imageBuf[idx] <= .8f) return;//Ignore weak thresholds at first
	
	//Make sure it doesn't loop back into an already covered path
	while(imageBuf[idx] != 0 && image[idx] == 0)
	{
		image[idx] = imageBuf[idx];
		if(angles[idx] == 0) idx += 1;
		else if(angles[idx] == 1) idx -= width - 1;
		else if(angles[idx] == 2) idx -= width;
		else if(angles[idx] == 3) idx += width - 1;
	}
	
	if(idx < 0 || idx > width * height) return;
}

int main(int argc, char* argv[])
{
	
	printf("%s\n", argv[0]);
	hipEvent_t start, stop;
	float timer;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Have the card compute the matrix for the gaussion blur
	float* d_gaussMat;
	float* h_gaussMat = (float*)malloc(sizeof(float) * gaussSize * gaussSize);
	hipMalloc((void**)&d_gaussMat, sizeof(float) * gaussSize * gaussSize);

	dim3 blockSize(gaussSize, gaussSize);
	dim3 gridSize(1,1);
	ComputeGuassian<<<gridSize, blockSize>>>(d_gaussMat, gaussSize);
	//hipDeviceSynchronize();
	hipMemcpy(h_gaussMat, d_gaussMat, sizeof(float) * gaussSize * gaussSize, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	for(int i = 0; i < gaussSize * gaussSize; i++)
	{
		printf("%f ", h_gaussMat[i]);
		if(i % 3 == 2) printf("\n");
	}
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	
	//Load an image
	BITMAPINFOHEADER bmpInfo;
	BITMAPFILEHEADER bitmapHeader;
	unsigned char* cImage = LoadBitmapFile("../Images/GT.bmp", &bitmapHeader, &bmpInfo);
	int imageSize = bmpInfo.biSizeImage;
	float *h_image, *d_image, *d_buf, *d_angles, *d_bw, *h_bw;

	h_image = (float*)malloc(sizeof(float) * imageSize);
	for(int i = 0; i < imageSize; i++)
		h_image[i] = (float)cImage[i] / 256.0f;
		
	//divide first to get the floor
	int sharedSize = sizeof(float) * (cuda_threadX + gaussSize) * (cuda_threadY + gaussSize);
	printf("Share size: %d\n", sharedSize);
	
	hipEventRecord(start, 0);
	hipMalloc((void**)&d_image, sizeof(float) * imageSize);
	hipMalloc((void**)&d_bw, sizeof(float) * imageSize / 3);
	hipMalloc((void**)&d_buf, sizeof(float) * imageSize / 3);
	hipMalloc((void**)&d_angles, sizeof(float) * imageSize / 3);
	hipMemcpy(d_image, h_image, sizeof(float) * imageSize, hipMemcpyHostToDevice);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timer, start, stop);
	printf("Load time: %f ms\n", timer);
	
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	
	//Convert the image into black and white
	hipEventRecord(start, 0);
	blockSize = dim3(cuda_threadX, cuda_threadY);
	gridSize = dim3(bmpInfo.biWidth / cuda_threadX, bmpInfo.biHeight / cuda_threadY);
	MakeBW<<<gridSize, blockSize>>>(d_image, d_bw, d_buf, d_angles, bmpInfo.biWidth, bmpInfo.biHeight);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timer, start, stop);
	printf("BW time: %f ms\n", timer);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	
	//Blur the image
	hipEventRecord(start, 0);
	GaussianBlur<<<gridSize, blockSize, sharedSize>>>();
	hipDeviceSynchronize();
	CopyToBuffer<<<gridSize, blockSize>>>();
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timer, start, stop);
	printf("Blur time: %f ms\n", timer);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	
	//Find the gradients
	hipEventRecord(start, 0);
	FindGradient<<<gridSize, blockSize, sharedSize>>>();
	hipDeviceSynchronize();
	CopyToBuffer<<<gridSize, blockSize>>>();
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timer, start, stop);
	printf("Gradient time: %f ms\n", timer);
	
	//Non-Maximum suppression
	hipEventRecord(start, 0);
	Suppression<<<gridSize, blockSize>>>();
	hipDeviceSynchronize();
	CopyToBuffer<<<gridSize, blockSize>>>();
	hipDeviceSynchronize();
	
	//hysteresis 
	/*hipEventRecord(start, 0);
	hysteresis<<<gridSize, blockSize>>>();
	hipDeviceSynchronize();*/
	
	h_bw = (float*)malloc(sizeof(float) * imageSize);
	hipMemcpy(h_bw, d_bw, sizeof(float) * imageSize / 3, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < imageSize; i++)
	{
		cImage[i] = (unsigned char)(h_bw[i / 3] * 256);
	}
	
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	
	SaveBitmapFile("GT.bmp", cImage, &bitmapHeader, &bmpInfo);
}
